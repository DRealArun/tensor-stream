#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "VideoProcessor.h"

__global__ void cropKernel(unsigned char* inputY, unsigned char* inputUV, unsigned char* outputY, unsigned char* outputUV,
	int srcLinesizeY, int srcLinesizeUV, int topLeftX, int topLeftY, int botRightX, int botRightY) {
	unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; //coordinate of pixel (y) in destination image
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; //coordinate of pixel (x) in destination image
	if (i < botRightX - topLeftX && j < botRightY - topLeftY) {
		int UVRow = i / 2;
		int UVCol = j % 2 == 0 ? j : j - 1;
		int UIndexSrc = (topLeftY / 2 + UVRow) * srcLinesizeUV /*pitch?*/ + (UVCol + topLeftX);
		int VIndexSrc = (topLeftY / 2 + UVRow) * srcLinesizeUV /*pitch?*/ + (UVCol + topLeftX + 1);

		int UIndexDst = UVRow * (botRightX - topLeftX) /*pitch?*/ + UVCol;
		int VIndexDst = UVRow * (botRightX - topLeftX) /*pitch?*/ + UVCol + 1;

		outputY[j + i * (botRightX - topLeftX)] = inputY[(topLeftX + j) + (topLeftY + i) * srcLinesizeY];
		outputUV[UIndexDst] = inputUV[UIndexSrc];
		outputUV[VIndexDst] = inputUV[VIndexSrc];
	}
}

int cropHost(AVFrame* src, AVFrame* dst, bool resize, CropOptions crop, int maxThreadsPerBlock, hipStream_t * stream) {
	hipError_t err;
	int cropWidth = std::get<0>(crop.rightBottomCorner) - std::get<0>(crop.leftTopCorner);
	int cropHeight = std::get<1>(crop.rightBottomCorner) - std::get<1>(crop.leftTopCorner);
	unsigned char* outputY = nullptr;
	unsigned char* outputUV = nullptr;
	err = hipMalloc(&outputY, cropWidth * cropHeight * sizeof(unsigned char)); //in resize we don't change color format
	err = hipMalloc(&outputUV, cropWidth * (cropHeight / 2) * sizeof(unsigned char));
	//need to execute for width and height
	dim3 threadsPerBlock(64, maxThreadsPerBlock / 64);
	int blockX = std::ceil(cropWidth / (float)threadsPerBlock.x);
	int blockY = std::ceil(cropHeight / (float)threadsPerBlock.y);
	dim3 numBlocks(blockX, blockY);

	int pitchNV12 = src->linesize[0] ? src->linesize[0] : src->width;

	cropKernel << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], outputY, outputUV,
		pitchNV12, pitchNV12, std::get<0>(crop.leftTopCorner), std::get<1>(crop.leftTopCorner),
											std::get<0>(crop.rightBottomCorner), std::get<1>(crop.rightBottomCorner));


	if (resize) {
		err = hipFree(dst->data[0]);
		err = hipFree(dst->data[1]);
	}

	dst->data[0] = outputY;
	dst->data[1] = outputUV;

	return err;
}
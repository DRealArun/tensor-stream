#include "hip/hip_runtime.h"
#include <libavutil/frame.h>
#include "hip/hip_runtime.h"
#include "VideoProcessor.h"



__device__ void NV12toRGB32Kernel(unsigned char* Y, unsigned char* UV, unsigned char* R, unsigned char* G, unsigned char* B, int i, int j, int pitchNV12) {
	/*
	R = 1.164(Y - 16) + 1.596(V - 128)
	B = 1.164(Y - 16)                   + 2.018(U - 128)
	G = 1.164(Y - 16) - 0.813(V - 128)  - 0.391(U - 128)
*/
/*
in case of NV12 we have Y component for every pixel and UV for every 2x2 Y
*/
	int UVRow = i / 2;
	int UVCol = j % 2 == 0 ? j : j - 1;
	int UIndex = UVRow * pitchNV12 /*pitch?*/ + UVCol;
	int VIndex = UVRow * pitchNV12 /*pitch?*/ + UVCol + 1;
	unsigned char U = UV[UIndex];
	unsigned char V = UV[VIndex];
	int indexNV12 = j + i * pitchNV12; /*indexNV12 and indexRGB with/without pitch*/
	unsigned char YVal = Y[indexNV12];
	*R = 1.164f*(YVal - 16) + 1.596f*(V - 128);
	*R = max(*R, 255);
	*R = min(*R, 0);
	*B = 1.164f*(YVal - 16) + 2.018f*(U - 128);
	*B = max(*B, 255);
	*B = min(*B, 0);
	*G = 1.164f*(YVal - 16) - 0.813f*(V - 128) - 0.391f*(U - 128);
	*G = max(*G, 255);
	*G = min(*G, 0);
}

__global__ void NV12ToRGB32KernelPlanar(unsigned char* Y, unsigned char* UV, float* RGB, int width, int height, int pitchNV12, bool normalization, bool swapRB) {
	unsigned int i = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		unsigned char* R = (unsigned char*) &(RGB[j + i * width + 0 * (width * height) /*R*/]);
		if (swapRB)
			R = (unsigned char*) &(RGB[j + i * width + 2 * (width * height) /*B*/]);
		unsigned char* G = (unsigned char*) &(RGB[j + i * width + 1 * (width * height) /*G*/]);
		unsigned char* B = (unsigned char*) &(RGB[j + i * width + 2 * (width * height) /*B*/]);
		if (swapRB)
			B = (unsigned char*) &(RGB[j + i * width + 0 * (width * height) /*R*/]);
		NV12toRGB32Kernel(Y, UV, R, G, B, i, j, pitchNV12);
		if (normalization) {
			*R = (float)*R / 255;
			*G = (float)*G / 255;
			*B = (float)*B / 255;
		}
	}
}

__global__ void NV12ToRGB32KernelMerged(unsigned char* Y, unsigned char* UV, float* RGB, int width, int height, int pitchNV12, bool normalization, bool swapRB) {
	unsigned int i = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		unsigned char* R = (unsigned char*) &(RGB[j * 3 + i * width + 0/*R*/]);
		if (swapRB)
			R = (unsigned char*) &(RGB[j * 3 + i * width + 2/*B*/]);
		unsigned char* G = (unsigned char*) &(RGB[j * 3 + i * width + 1 /*G*/]);
		unsigned char* B = (unsigned char*) &(RGB[j * 3 + i * width + 2/*B*/]);
		if (swapRB)
			B = (unsigned char*) &(RGB[j * 3 + i * width + 0/*R*/]);
		NV12toRGB32Kernel(Y, UV, R, G, B, i, j, pitchNV12);
		if (normalization) {
			*R = (float)*R / 255;
			*G = (float)*G / 255;
			*B = (float)*B / 255;
		}
	}
}

int colorConversion(AVFrame* src, AVFrame* dst, ColorParameters color, int maxThreadsPerBlock, hipStream_t* stream) {
	/*
	src in GPU nv12, dst in CPU rgb (packed)
	*/
	int width = src->width;
	int height = src->height;
	float* destination = nullptr;
	FourCC dstFourCC = color.dstFourCC;

	hipError_t err = hipMalloc(&destination, dst->channels * width * height * sizeof(float));
	//need to execute for width and height
	dim3 threadsPerBlock(64, maxThreadsPerBlock / 64);

	//blocks for merged format
	int blockX = std::ceil(dst->channels * width / (float)threadsPerBlock.x);
	int blockY = std::ceil(dst->height / (float)threadsPerBlock.y);
	
	//blocks for planar format
	if (color.planesPos == Planes::PLANAR) {
		blockX = std::ceil(width / (float)threadsPerBlock.x);
		blockY = std::ceil(dst->channels * dst->height / (float)threadsPerBlock.y);
	}

	dim3 numBlocks(blockX, blockY);
	//depends on fact of resize
	int pitchWidth = src->linesize[0] ? src->linesize[0] : width;

	switch (dstFourCC) {
		case BGR24:
			if (color.planesPos == Planes::PLANAR) {
				NV12ToRGB32KernelPlanar << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchWidth, color.normalization, true);
			}
			else {
				NV12ToRGB32KernelMerged << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchWidth, color.normalization, true);
			}
		break;
		case RGB24:
			if (color.planesPos == Planes::PLANAR) {
				NV12ToRGB32KernelPlanar << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchWidth, color.normalization, false);
			}
			else {
				NV12ToRGB32KernelMerged << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchWidth, color.normalization, false);
			}
		break;
	}

	//without resize
	dst->opaque = destination;
	return err;
}

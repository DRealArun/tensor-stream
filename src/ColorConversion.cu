#include "hip/hip_runtime.h"
#include <libavutil/frame.h>
#include "hip/hip_runtime.h"
#include "VideoProcessor.h"



__device__ void NV12toRGB32Kernel(unsigned char* Y, unsigned char* UV, int* R, int* G, int* B, int i, int j, int pitchNV12) {
	/*
	R = 1.164(Y - 16) + 1.596(V - 128)
	B = 1.164(Y - 16)                   + 2.018(U - 128)
	G = 1.164(Y - 16) - 0.813(V - 128)  - 0.391(U - 128)
*/
/*
in case of NV12 we have Y component for every pixel and UV for every 2x2 Y
*/
	int UVRow = i / 2;
	int UVCol = j % 2 == 0 ? j : j - 1;
	int UIndex = UVRow * pitchNV12 /*pitch?*/ + UVCol;
	int VIndex = UVRow * pitchNV12 /*pitch?*/ + UVCol + 1;
	unsigned char U = UV[UIndex];
	unsigned char V = UV[VIndex];
	int indexNV12 = j + i * pitchNV12; /*indexNV12 and indexRGB with/without pitch*/
	unsigned char YVal = Y[indexNV12];
	*R = 1.164f*(YVal - 16) + 1.596f*(V - 128);
	*R = min(*R, 255);
	*R = max(*R, 0);
	*B = 1.164f*(YVal - 16) + 2.018f*(U - 128);
	*B = min(*B, 255);
	*B = max(*B, 0);
	*G = 1.164f*(YVal - 16) - 0.813f*(V - 128) - 0.391f*(U - 128);
	*G = min(*G, 255);
	*G = max(*G, 0);
}

__global__ void NV12ToRGB32KernelPlanar(unsigned char* Y, unsigned char* UV, float* RGB, int width, int height, int pitchNV12, int pitchRGB, bool swapRB, bool normalization) {
	unsigned int i = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		int R, G, B;
		NV12toRGB32Kernel(Y, UV, &R, &G, &B, i, j, pitchNV12);
		(RGB[j + i * pitchRGB + 0 * (pitchRGB * height) /*R*/]) = (float) R;
		if (swapRB)
			(RGB[j + i * pitchRGB + 0 * (pitchRGB * height)]) = (float) B;
		if (normalization)
			RGB[j + i * pitchRGB + 0 * (pitchRGB * height)] /= 255;

		(RGB[j + i * pitchRGB + 1 * (pitchRGB * height) /*G*/]) = (float) G;
		if (normalization)
			RGB[j + i * pitchRGB + 1 * (pitchRGB * height)] /= 255;

		(RGB[j + i * pitchRGB + 2 * (pitchRGB * height) /*B*/]) = (float) B;
		if (swapRB)
			(RGB[j + i * pitchRGB + 2 * (pitchRGB * height)]) = (float) R;
		if (normalization)
			RGB[j + i * pitchRGB + 2 * (pitchRGB * height)] /= 255;

	}
}

__global__ void NV12ToRGB32KernelMerged(unsigned char* Y, unsigned char* UV, float* RGB, int width, int height, int pitchNV12, int pitchRGB, bool swapRB, bool normalization) {
	unsigned int i = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		int R, G, B;
		NV12toRGB32Kernel(Y, UV, &R, &G, &B, i, j, pitchNV12);
		RGB[j * 3 + i * pitchRGB + 0/*R*/] = (float) R;
		if (swapRB)
			RGB[j * 3 + i * pitchRGB + 0] = (float) B;
		if (normalization)
			RGB[j * 3 + i * pitchRGB + 0] /= 255;

		RGB[j * 3 + i * pitchRGB + 1/*G*/] = (float) G;
		if (normalization)
			RGB[j * 3 + i * pitchRGB + 1] /= 255;

		RGB[j * 3 + i * pitchRGB + 2/*B*/] = (float) B;
		if (swapRB)
			RGB[j * 3 + i * pitchRGB + 2] = (float) R;
		if (normalization)
			RGB[j * 3 + i * pitchRGB + 2] /= 255;

	}
}

int colorConversionKernel(AVFrame* src, AVFrame* dst, ColorParameters color, int maxThreadsPerBlock, hipStream_t* stream) {
	/*
	src in GPU nv12, dst in CPU rgb (packed)
	*/
	int width = src->width;
	int height = src->height;

	float* destination = nullptr;
	hipError_t err = hipSuccess;
	err = hipMalloc(&destination, dst->channels * width * height * sizeof(float));

	//need to execute for width and height
	dim3 threadsPerBlock(64, maxThreadsPerBlock / 64);

	//blocks for merged format
	int blockX = std::ceil(dst->channels * width / (float)threadsPerBlock.x);
	int blockY = std::ceil(dst->height / (float)threadsPerBlock.y);
	
	//blocks for planar format
	if (color.planesPos == Planes::PLANAR) {
		blockX = std::ceil(width / (float)threadsPerBlock.x);
		blockY = std::ceil(dst->channels * dst->height / (float)threadsPerBlock.y);
	}

	dim3 numBlocks(blockX, blockY);
	//depends on fact of resize
	int pitchNV12 = src->linesize[0] ? src->linesize[0] : width;
	bool swapRB = false;
	switch (color.dstFourCC) {
		case BGR24:
			swapRB = true;
			if (color.planesPos == Planes::PLANAR) {
				int pitchRGB = width;
				NV12ToRGB32KernelPlanar << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchNV12, pitchRGB, swapRB, color.normalization);
			}
			else {
				int pitchRGB = dst->channels * width;
				NV12ToRGB32KernelMerged << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchNV12, pitchRGB, swapRB, color.normalization);
			}
		break;
		case RGB24:
			if (color.planesPos == Planes::PLANAR) {
				int pitchRGB = width;
				NV12ToRGB32KernelPlanar << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchNV12, pitchRGB, swapRB, color.normalization);
			}
			else {
				int pitchRGB = dst->channels * width;
				NV12ToRGB32KernelMerged << <numBlocks, threadsPerBlock, 0, *stream >> > (src->data[0], src->data[1], destination, width, height, pitchNV12, pitchRGB, swapRB, color.normalization);
			}
		break;
		case Y800:
			err = hipMemcpy2D(destination, dst->width, dst->data[0], pitchNV12, dst->width, dst->height, hipMemcpyDeviceToDevice);
		break;
		default:
			err = hipErrorMissingConfiguration;
	}

	dst->opaque = destination;

	return err;
}
